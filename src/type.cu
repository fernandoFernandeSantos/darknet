#include "hip/hip_runtime.h"
/*
 * type.c
 *
 *  Created on: 13/09/2018
 *      Author: fernando
 */

/**
 * Make real_t3 type
 * if cuda is activated it must be a function
 * accessible bye host or device
 */

#include "type.h"

#ifdef GPU
#include "hip/hip_runtime.h"
#endif
/**
 * Transform a float array into an half precision
 */

__global__ void float_to_half_array(real_t_device* dst, float* src,
		size_t size) {
	int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
	if (i < size)
		dst[i] = __float2half(src[i]);
}

void transform_float_to_half_array(real_t_device* dst, float* src, size_t n) {
	float_to_half_array<<<cuda_gridsize(n), BLOCK>>>(dst, src, n);
	check_error(hipPeekAtLastError());

}

/**
 * Read a file for all precisions
 */
int fread_float_to_real_t(real_t* dst, size_t siz, size_t times, FILE* fp) {
	float* temp = (float*) calloc(times, sizeof(float));
	if (temp == NULL) {
		return -1;
	}
	int fread_result = fread(temp, sizeof(float), times, fp);
	if (fread_result != times) {
		free(temp);
		return -1;
	}
	int i;
	for (i = 0; i < times; i++) {
		//TODO: make ready for half
		dst[i] = real_t(temp[i]);
	}
	free(temp);
	return fread_result;

}

inline int cuda_get_device() {
	int n = 0;
	hipError_t status = hipGetDevice(&n);
	check_error(status);
	return n;
}

inline hipStream_t get_cuda_stream() {
	static hipStream_t streamsArray[16]; // hipStreamSynchronize( get_cuda_stream() );
	static int streamInit[16] = { 0 };

	int i = cuda_get_device();
	if (!streamInit[i]) {
		hipError_t status = hipStreamCreate(&streamsArray[i]);
		//hipError_t status = hipStreamCreateWithFlags(&streamsArray[i], hipStreamNonBlocking);
		if (status != hipSuccess) {
			printf(" hipStreamCreate error: %d \n", status);
			const char *s = hipGetErrorString(status);
			char buffer[256];
			printf("CUDA Error: %s\n", s);
			status = hipStreamCreateWithFlags(&streamsArray[i],
					hipStreamDefault);
			check_error(status);
		}
		streamInit[i] = 1;
	}
	return streamsArray[i];
}

FP16Array::FP16Array(size_t size, float* fp32_array) {
	printf("passou no fp16array");
	hipError_t status = hipMalloc((void**) this->fp16_ptr,
			sizeof(real_t_fp16) * size);
	printf("aqui agoraz\n");
	check_error(status);

	this->fp32_ptr = fp32_array;
	this->size = size;
}

FP16Array::~FP16Array() {
	if (this->fp16_ptr != nullptr) {
		hipError_t status = hipFree(this->fp16_ptr);
		check_error(status);
	}
}

__global__ void cuda_f32_to_f16(real_t_device* input_f32, size_t size,
		real_t_fp16 *output_f16) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		output_f16[idx] = __float2half(input_f32[idx]);
	//if (idx < size) *((unsigned short *)output_f16 + idx) = __float2half(input_f32[idx]);
}

void FP16Array::cuda_convert_f32_to_f16() {
	cuda_f32_to_f16<<<size / BLOCK + 1, BLOCK, 0, get_cuda_stream()>>>(
			this->fp32_ptr, size, this->fp16_ptr);
}

__global__ void cuda_f16_to_f32(real_t_fp16* input_f16, size_t size,
		float *output_f32) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
		output_f32[idx] = __half2float(input_f16[idx]);
}

void FP16Array::cuda_convert_f16_to_f32() {
	cuda_f16_to_f32<<<this->size / BLOCK + 1, BLOCK, 0, get_cuda_stream()>>>(
			this->fp16_ptr, size, this->fp32_ptr);
}

